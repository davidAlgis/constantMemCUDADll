#include "hip/hip_runtime.h"
#include "cstMem.cuh"

namespace dllA
{

__constant__ float cstMem;

__device__ __host__ float *getCstMem()
{
    return &cstMem;
}

__global__ void retrieveCstMem(float* output)
{
    *output = *getCstMem();
}

float kernelCallerRetrieveCstMem()
{
    float* d_output;
    float* h_output = new float();
    CUDA_CHECK(hipMalloc(&d_output, sizeof(float)));
    
    retrieveCstMem<<<1,1>>>(d_output);

    CUDA_CHECK(hipMemcpy(h_output,d_output, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_output));
    float ret = *h_output;
    delete(h_output);
    return ret;
}
} // namespace dllA